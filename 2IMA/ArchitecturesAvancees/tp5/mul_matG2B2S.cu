 
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>

//une tuile carre
#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y BLOCK_SIZE_X 
#define TILE_SIZE BLOCK_SIZE_X 

// CUDA Kernel
__global__ void
matrixMul( float* C, float* A, float* B, int wA)
{
 
   // Coordonnees du thread
   // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
 
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
 
    // indice de la premiere tuile de A traitee par le bloc
    int a1 = wA * BLOCK_SIZE_Y * by;
 
    // indice de la derniere tuile de A traitee par le bloc
    int a2   = a1 + wA - TILE_SIZE;
 
    // pas d'adresse entre tuiles de la matrice A
    int apas  = TILE_SIZE;
 
    // indice de la premiere tuile de B traitee par le bloc
    int b1 = BLOCK_SIZE_X * bx;
 
    // pas d'adresse entre tuiles de la matrice B
    int bpas  = TILE_SIZE * wA;

    float Csub=0.0;

    // boucle sur les tuiles
    for (int a = a1, b = b1;
             a <= a2;
             a += apas, b += bpas) 
    {

        // Declaration en memoire partagee de la matrice devant contenir une tuie de A
        __shared__ float As[BLOCK_SIZE_Y][TILE_SIZE];
 
        // Declaration en memoire partagee de la matrice devant contenir une tuie de A
        __shared__ float Bs[TILE_SIZE][BLOCK_SIZE_X];
 
        // Copie des tuiles depuis la memoire globale vers la memoire partagee
        // chaque thread copie un element de chaque matrice
	  As[ty][tx] = A[a + wA * ty + tx];
	  Bs[ty][tx] = B[b + wA * ty + tx];

        // Synchronisation pour que les threads attendent la fin de la copie 
        __syncthreads();
 
        // multiplication entre tuiles 
        for ( int k = 0; k < TILE_SIZE; k +=1) {
            Csub += As[ty][k] * Bs[k][tx];
		}
 
         // Synchronisation pour que les threads attendent la fin du calcul
        __syncthreads();
 
    }
 
    // Enregistrement du resultat
    // indice de la ligne 
    C[a1 + wA * ty + b1 + tx] = Csub;
}
  
/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int
main(int argc, char** argv)
{
	int i, j;
	int TM=2048;
	//BLOCK_SIZE_X =BLOCK_SIZE;
	//BLOCK_SIZE_Y = BLOCK_SIZE;
	if (argc>1) {
		 TM=atoi(argv[1]);
     }

	// Verification de la bonne taille par rapport aux dimensions des blocs
	if ((TM % BLOCK_SIZE_X) !=0) {
		printf("Taille matrice non multiple de taille bloc %d \n", BLOCK_SIZE_X);
		exit(1);
	}
	if ((TM % BLOCK_SIZE_Y) !=0) {
		printf("Taille matrice non multiple de taille bloc %d \n", BLOCK_SIZE_Y);
		exit(1);
	}
     // 1. Allocation memoire sur CPU
   unsigned int size_A = TM*TM;
   unsigned int mem_size_A = sizeof(float) * size_A;
   float* h_A = (float*) malloc(mem_size_A);
 
   unsigned int size_B = TM*TM;
   unsigned int mem_size_B = sizeof(float) * size_B;
   float* h_B = (float*) malloc(mem_size_B);
 
   unsigned int size_C = TM*TM;
   unsigned int mem_size_C = sizeof(float) * size_C;
   float* h_C = (float*) malloc(mem_size_C);

   // 2. initialisation des donnees CPU
   // initialisation des matrices avec des valeurs permettant de verifier le resultat
   for(i = 0; i < TM; i++){
      for(j = 0; j < TM; j++){
      	h_A[i*TM+j] = 1.0;
      	h_B[i*TM+j] = 1.0;
		h_C[i*TM+j] = 0.0;

		if (i==j) {
			h_A[i*TM+j]=(float) (i+1);
			h_B[i*TM+j]=(float) (i+1);
		}
      }
   }

 
   // 3. allocation memoire sur GPU
   float* d_A;
   float* d_B;
   float* d_C;
   hipMalloc((void**) &d_A, mem_size_A);
   hipMalloc((void**) &d_B, mem_size_B);
   hipMalloc((void**) &d_C, mem_size_C);
 
   // 4. copie des donnes CPU vers GPU

   // mesure du temps 
   float  elapsedTime ;
   hipEvent_t start , stop ;
   hipEventCreate (&start ) ;
   hipEventCreate (&stop ) ;
   hipEventRecord ( start , 0 ) ;

   hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

   // 5. choix de la structure : grille et blocs
   dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
   dim3 grid(TM / threads.x, TM / threads.y);
 
   printf("bloc %d %d \n", BLOCK_SIZE_X, BLOCK_SIZE_Y);
   printf("grille %d %d \n", TM / threads.x, TM / threads.y);

   // 6. execution du kernel
   matrixMul<<< grid, threads >>>(d_C, d_A, d_B, TM);

   // 7. copie du resultat depuis le GPU
   hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

   hipEventRecord ( stop , 0 ) ;
   hipEventSynchronize ( stop ) ;
   hipEventElapsedTime ( &elapsedTime , start , stop ) ;
   hipEventDestroy ( start ) ;
   hipEventDestroy ( stop ) ;
   printf ( "Temps consomme: %f secondes\n" , elapsedTime / 1000.0 ) ;
 
  // Verification des resultats

  for(i = 0; i < TM; i++){
    for(j = 0; j < TM; j++){
	if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) 	   	{
		printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); 
		exit(1);
		}
		else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))) 			{
			printf("Erreur i: %d j: %d\n", i, j);
			exit(1);
		}
      }
   } 

   // 8. liberation de la memoire
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C); 
}
