#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


// CUDA Kernel
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
    // calcul des coordonnees du thread
    int i =  blockDim.y * blockIdx.y + threadIdx.y;
    int j =  blockDim.x * blockIdx.x + threadIdx.x;

    // calcul de c[i][j]
    float cs = 0;
    for (int k = 0; k < TM; k++) {
        cs += A[i*TM + k] * B[k*TM + j];
    }
    C[i*TM + j] = cs;

}


/////////////////////////////////////////////////////////
// Programme main
/////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
    int i, j, GRID_SIZE_X, GRID_SIZE_Y, BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_DIM, DEC, TAILLE;
    int TM=2048;
    hipError_t cerror;

    if (argc>1) {
        TM=atoi(argv[1]);
    }
    DEC = 1;
    BLOCK_DIM = 512;
    BLOCK_SIZE_X = BLOCK_DIM/DEC; 
    BLOCK_SIZE_Y = BLOCK_DIM/BLOCK_SIZE_X; 
    GRID_SIZE_X = TM/BLOCK_SIZE_X;
    GRID_SIZE_Y = TM;
    TAILLE = TM*TM * sizeof(float);

    // definiton de la grille et des blocs
    dim3 block(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid(GRID_SIZE_X, GRID_SIZE_Y);

    // allocation des matrices sur CPU
    float *h_A, *h_B, *h_C ;
    h_A = (float*) malloc(TAILLE);
    h_B = (float*) malloc(TAILLE);
    h_C = (float*) malloc(TAILLE);

    // initialisation des matrices avec des valeurs permettant de verifier le resultat
    for(i = 0; i < TM; i++){
        for(j = 0; j < TM; j++){
            h_A[i*TM+j] = 1.0;
            h_B[i*TM+j] = 1.0;
            h_C[i*TM+j] = 0.0;

            if (i==j) {
                h_A[i*TM+j]=(float) (i+1);
                h_B[i*TM+j]=(float) (i+1);
            }
        }
    }

    // Pour mesurer le temps de calcul
    float tc;
    hipEvent_t depart, arret;
    hipEventCreate(&depart);
    hipEventCreate(&arret);
    hipEventRecord(depart,0);

    // allocation des matrices sur GPU
    float *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, TAILLE);
    hipMalloc((void**) &d_B, TAILLE);
    hipMalloc((void**) &d_C, TAILLE);

    // copie des matrives A et B depuis le CPU vers le GPU
    hipMemcpy(d_A, h_A, TAILLE, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, TAILLE, hipMemcpyHostToDevice);

    // lancement des threads
    matrixMul<<< grid, block >>>(d_C, d_A, d_B, TM);

    // Vérification en cas d'erreur
    cerror=hipGetLastError(); 
    if ((int)cerror !=0) { 
        printf("Erreur appel kernel %d \n", (int) cerror);
        exit(cerror);
    }
    // copie de la matrive C depuis le GPU
    hipMemcpy(h_C, d_C, TAILLE, hipMemcpyDeviceToHost);

    // Mesure du temps : transfert + calcul
    hipEventRecord(arret,0);
    hipEventSynchronize(arret);
    hipEventElapsedTime(&tc,depart, arret);

    hipEventDestroy(depart);
    hipEventDestroy(arret);
    printf("Temps consommé : %f seconde\n", tc/1000.0);

    // verification du resultat
    for(i = 0; i < TM; i++){
        for(j = 0; j < TM; j++){
            if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) {
                printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] );
                exit(1);
            } else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))) {
                printf("Erreur i: %d j: %d\n", i, j);
                exit(1);
            }
        }
    } 

    // liberation de la memoire
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0; 
}
