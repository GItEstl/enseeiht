#include "hip/hip_runtime.h"
 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
 

// CUDA Kernel
__global__ void matrixMul( float* C, float* A, float* B, int TM)
{
   // calcul des coordonnees du thread
   //int i =  ;
   //int j =  ;
 
   // calcul de c[i][j]
 
   // ....

}
 
 
/////////////////////////////////////////////////////////
// Programme main
/////////////////////////////////////////////////////////
 
int main(int argc, char** argv)
{
	int i, j, GRID_SIZE_X, BLOCK_SIZE_X;
	int TM=1024;
	hipError_t cerror;

	if (argc>1) {
		 TM=atoi(argv[1]);
	}
	BLOCK_SIZE_X = TM; 
	GRID_SIZE_X = TM;
 
   // definiton de la grille et des blocs
   //dim3 block ...;
   //dim3 grid ...;

   // allocation des matrices sur CPU
   float *h_A, *h_B, *h_C ;
   //.....
   
   // initialisation des matrices avec des valeurs permettant de verifier le resultat
   for(i = 0; i < TM; i++){
      for(j = 0; j < TM; j++){
      	h_A[i*TM+j] = 1.0;
      	h_B[i*TM+j] = 1.0;
		h_C[i*TM+j] = 0.0;

		if (i==j) {
			h_A[i*TM+j]=(float) (i+1);
			h_B[i*TM+j]=(float) (i+1);
		}
      }
   }

   // Pour mesurer le temps de calcul
   float tc;
   hipEvent_t depart, arret;
   hipEventCreate(&depart);
   hipEventCreate(&arret);
   hipEventRecord(depart,0);

   // allocation des matrices sur GPU
   float *d_A, *d_B, *d_C;
   //...
 
   // copie des matrives A et B depuis le CPU vers le GPU
   //.....

   // lancement des threads
   matrixMul<<< grid, block >>>(d_C, d_A, d_B, TM);
 
   // Vérification en cas d'erreur
   cerror=hipGetLastError(); 
   if ((int)cerror !=0) { 
     printf("Erreur appel kernel %d \n", (int) cerror);
     exit(cerror);
   }
   // copie de la matrive C depuis le GPU
   // ....

   // Mesure du temps : transfert + calcul
   hipEventRecord(arret,0);
   hipEventSynchronize(arret);
   hipEventElapsedTime(&tc,depart, arret);

   hipEventDestroy(depart);
   hipEventDestroy(arret);
   printf("Temps consommé : %f seconde\n", tc/1000.0);

   // verification du resultat
  for(i = 0; i < TM; i++){
    for(j = 0; j < TM; j++){
	if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) 	   	{
		printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); exit(1);
		}
		else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))) 			{
			printf("Erreur i: %d j: %d\n", i, j);
			exit(1);
		}
      }
   } 

   // liberation de la memoire
   free(h_A);
   free(h_B);
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
 
}
