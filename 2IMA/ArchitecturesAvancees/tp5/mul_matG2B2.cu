
#include <stdio.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32
#define TILE_SIZE BLOCK_SIZE_X
#define LOOP_DIVIDER 8

// CUDA Kernel
    __global__ void
matrixMul( float* C, float* A, float* B, int wA)
{

    // Coordonnees absolues du thread

    int i = blockIdx.y * blockDim.y+ threadIdx.y;
    int j = blockIdx.x * blockDim.x+ threadIdx.x;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // chque thread calcule C[i][j]
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];
    float Csub=0.0;
    for (int ke = 0; ke < wA; ke += TILE_SIZE) {
        As[ty][tx] = A[i * wA + (ke+tx)];
        Bs[ty][tx] = B[(ke+ty) * wA + j];

        __syncthreads();
        for (int k = 0; k < TILE_SIZE; k++) {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }
    C[i * wA + j] = Csub;
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////

int main(int argc, char** argv)
{
    int i, j;
    int TM=2048;
    // TM peut etre saisie comme argument de la commande
    if (argc>1) {
        TM=atoi(argv[1]);
    }

    // Verification de la bonne taille par rapport aux dimensions des blocs
    if ((TM % BLOCK_SIZE_X) !=0) {
        printf("Taille matrice non multiple de taille bloc %d \n", BLOCK_SIZE_X);
        exit(1);
    }
    if ((TM % BLOCK_SIZE_Y) !=0) {
        printf("Taille matrice non multiple de taille bloc %d \n", BLOCK_SIZE_Y);
        exit(1);
    }
    // 1. Allocation memoire sur CPU
    unsigned int size_A = TM*TM;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    unsigned int size_B = TM*TM;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    unsigned int size_C = TM*TM;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* h_C = (float*) malloc(mem_size_C);

    // 2. initialisation des donnees CPU
    // initialisation des matrices avec des valeurs permettant de verifier le resultat
    for(i = 0; i < TM; i++){
        for(j = 0; j < TM; j++){
            h_A[i*TM+j] = 1.0;
            h_B[i*TM+j] = 1.0;
            h_C[i*TM+j] = 0.0;

            if (i==j) {
                h_A[i*TM+j]=(float) (i+1);
                h_B[i*TM+j]=(float) (i+1);
            }
        }
    }


    // 3. allocation memoire sur GPU
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
    hipMalloc((void**) &d_C, mem_size_C);

    // 4. copie des donnes CPU vers GPU

    // mesure du temps de transfert
    float  elapsedTime ;
    hipEvent_t start , stop ;
    hipEventCreate (&start ) ;
    hipEventCreate (&stop ) ;
    hipEventRecord ( start , 0 ) ;

    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);


    // 5. choix de la structure : grille et blocs
    dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid(TM / threads.x, TM / threads.y);

    printf("bloc %d %d \n", BLOCK_SIZE_X, BLOCK_SIZE_Y);
    printf("grille %d %d \n", TM / threads.x, TM / threads.y);

    // 6. execution du kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, TM);

    // 7. copie du resultat depuis le GPU
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    hipEventRecord ( stop , 0 ) ;
    hipEventSynchronize ( stop ) ;
    hipEventElapsedTime ( &elapsedTime , start , stop ) ;
    hipEventDestroy ( start ) ;
    hipEventDestroy ( stop ) ;
    printf ( "Temps consomme: %f secondes\n" , elapsedTime / 1000.0 ) ;

    // Verification des resultats

    for(i = 0; i < TM; i++){
        for(j = 0; j < TM; j++){
            if ((i==j) && (h_C[i*TM+j] != (float)((i+1)*(i+1)+TM-1))) 	   	{
                printf("Erreur i: %d j: %d %f\n", i, j, h_C[i*TM+j] ); 
                exit(1);
            }
            else if ((i!=j) && (h_C[i*TM+j] != (float)(i+j+TM))) 			{
                printf("Erreur i: %d j: %d\n", i, j);
                exit(1);
            }
        }
    } 

    // 8. liberation de la memoire
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
